#include "hip/hip_runtime.h"
//
// Created by alexeymar on 29-Nov-19.
//

#include "spdk_reader.h"
#include <iostream>

const size_t chars_to_print = 200;

__global__
void print_gpu_mem(char *c, int n) {
	for (int i = 0; i < n; i++) {
		printf("%c", c[i]);
	}
	printf("\n");
}

int main(int argc, char *argv[]) {
	if (argc < 3) {
		std::cerr << "Specify controller address and file path" << std::endl;
		return 1;
	}
	std::cout << "Working with controller " << argv[1] << std::endl;
	std::cout << "File " << argv[2] << std::endl;
	try {
		spdk_reader_ctx reader{argv[1]};
		auto alloc_size = reader.get_aligned_file_size(argv[2]);

		std::cout << "Read tp CPU" << std::endl;
		auto cpu_buffer = spdk_reader_ctx::get_cpu_mem(alloc_size);
		int rc = reader.do_read(argv[2], cpu_buffer.get());
		if (rc) {
			std::cerr << "Read completed with error " << rc << std::endl;
			return 1;
		}
		std::cout << "Print first " << chars_to_print << "  characters" << std::endl;
		for (uint32_t i = 0; i < chars_to_print && i < alloc_size; i++) {
			std::cout << static_cast<char *>(cpu_buffer.get())[i];
		}
		std::cout << std::endl;

		std::cout << "Read to GPU memory" << std::endl;
		auto gpu_buffer = spdk_reader_ctx::get_gpu_mem(alloc_size);
		rc = reader.do_read(argv[2], gpu_buffer.get());
		if (rc) {
			std::cerr << "Read completed with error " << rc << std::endl;
			return 1;
		}
		std::cout << "Print first " << chars_to_print << "  characters" << std::endl;
		print_gpu_mem << < 1, 1 >> > (static_cast<char *>(gpu_buffer.get()),
			std::min(chars_to_print, alloc_size));

	} catch (std::runtime_error &e) {
		std::cerr << "Exception caught: " << e.what() << std::endl;
	}
}