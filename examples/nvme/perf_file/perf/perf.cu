#include "hip/hip_runtime.h"
/*-
 *   BSD LICENSE
 *
 *   Copyright (c) Intel Corporation.
 *   All rights reserved.
 *
 *   Copyright (c) 2019 Mellanox Technologies LTD. All rights reserved.
 *
 *   Redistribution and use in source and binary forms, with or without
 *   modification, are permitted provided that the following conditions
 *   are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in
 *       the documentation and/or other materials provided with the
 *       distribution.
 *     * Neither the name of Intel Corporation nor the names of its
 *       contributors may be used to endorse or promote products derived
 *       from this software without specific prior written permission.
 *
 *   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *   "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *   LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 *   A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 *   OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 *   SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 *   LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 *   DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 *   THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 *   (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 *   OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "spdk/stdinc.h"

#include "spdk/env.h"
#include "spdk/fd.h"
#include "spdk/nvme.h"
#include "spdk/vmd.h"
#include "spdk/queue.h"
#include "spdk/string.h"
#include "spdk/nvme_intel.h"
#include "spdk/histogram_data.h"
#include "spdk/endian.h"
#include "spdk/dif.h"
#include "spdk/util.h"
#include "spdk/log.h"
#include "spdk/likely.h"

#if HAVE_LIBAIO
#include <libaio.h>
#endif

#include <sys/fcntl.h>
#include <sys/ioctl.h>
#include <linux/fs.h>
#include <linux/fiemap.h>
#include <infiniband/verbs.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <rdma/rdma_cma.h>

struct ctrlr_entry {
	struct spdk_nvme_ctrlr			*ctrlr;
	enum spdk_nvme_transport_type		trtype;
	struct spdk_nvme_intel_rw_latency_page	*latency_page;

	struct spdk_nvme_qpair			**unused_qpairs;

	struct ctrlr_entry			*next;
	char					name[1024];
};

enum entry_type {
	ENTRY_TYPE_NVME_NS,
	ENTRY_TYPE_AIO_FILE,
};

struct ns_fn_table;

struct ns_entry {
	enum entry_type		type;
	const struct ns_fn_table	*fn_table;

	union {
		struct {
			struct spdk_nvme_ctrlr	*ctrlr;
			struct spdk_nvme_ns	*ns;
		} nvme;
#if HAVE_LIBAIO
		struct {
			int			fd;
		} aio;
#endif
	} u;

	struct ns_entry		*next;
	uint32_t		io_size_blocks;
	uint32_t		num_io_requests;
	uint64_t		size_in_ios;
	uint32_t		block_size;
	uint32_t		md_size;
	bool			md_interleave;
	bool			pi_loc;
	enum spdk_nvme_pi_type	pi_type;
	uint32_t		io_flags;
	char			name[1024];
};


struct ns_fn_table {
	void	(*setup_payload)(struct perf_task *task, uint8_t pattern);

	int	(*submit_io)(struct perf_task *task, struct ns_worker_ctx *ns_ctx,
			     struct ns_entry *entry, uint64_t offset_in_ios);

	void	(*check_io)(struct ns_worker_ctx *ns_ctx);

	void	(*verify_io)(struct perf_task *task, struct ns_entry *entry);

	int	(*init_ns_worker_ctx)(struct ns_worker_ctx *ns_ctx);

	void	(*cleanup_ns_worker_ctx)(struct ns_worker_ctx *ns_ctx);
};

static int g_outstanding_commands;

static bool g_latency_ssd_tracking_enable = false;
static int g_latency_sw_tracking_level = 0;

static bool g_vmd = false;

static struct ctrlr_entry *g_controllers = NULL;
static struct ns_entry *g_namespaces = NULL;
static int g_num_namespaces = 0;
static struct worker_thread *g_workers = NULL;
static int g_num_workers = 0;
static uint32_t g_master_core;

static uint64_t g_tsc_rate;

static uint32_t g_io_align = 0x200;
static uint32_t g_io_size_bytes;
static uint32_t g_max_io_md_size;
static uint32_t g_max_io_size_blocks;
static uint32_t g_metacfg_pract_flag;
static uint32_t g_metacfg_prchk_flags;
static int g_rw_percentage;
static int g_is_random;
static int g_queue_depth;
static int g_nr_io_queues_per_ns = 1;
static int g_nr_unused_io_queues = 0;
static int g_time_in_sec;
static uint32_t g_max_completions;
static int g_dpdk_mem;
static int g_shm_id = -1;
static uint32_t g_disable_sq_cmb;
static bool g_no_pci;
static bool g_warn;
static bool g_header_digest;
static bool g_data_digest;
static bool g_no_shn_notification = false;
static uint32_t g_keep_alive_timeout_in_ms = 0;

static const char *g_core_mask;

struct trid_entry {
	struct spdk_nvme_transport_id	trid;
	uint16_t			nsid;
	TAILQ_ENTRY(trid_entry)		tailq;
};

static TAILQ_HEAD(, trid_entry) g_trid_list = TAILQ_HEAD_INITIALIZER(g_trid_list);

static int g_aio_optind; /* Index of first AIO filename in argv */


static void
build_nvme_name(char *name, size_t length, struct spdk_nvme_ctrlr *ctrlr)
{
	const struct spdk_nvme_transport_id *trid;

	trid = spdk_nvme_ctrlr_get_transport_id(ctrlr);

	switch (trid->trtype) {
	case SPDK_NVME_TRANSPORT_PCIE:
		snprintf(name, length, "PCIE (%s)", trid->traddr);
		break;
	case SPDK_NVME_TRANSPORT_RDMA:
		snprintf(name, length, "RDMA (addr:%s subnqn:%s)", trid->traddr, trid->subnqn);
		break;
	case SPDK_NVME_TRANSPORT_TCP:
		snprintf(name, length, "TCP  (addr:%s subnqn:%s)", trid->traddr, trid->subnqn);
		break;
	default:
		fprintf(stderr, "Unknown transport type %d\n", trid->trtype);
		break;
	}
}

static void
register_ns(struct spdk_nvme_ctrlr *ctrlr, struct spdk_nvme_ns *ns)
{
	struct ns_entry *entry;
	const struct spdk_nvme_ctrlr_data *cdata;
	uint32_t max_xfer_size, entries, sector_size;
	uint64_t ns_size;
	struct spdk_nvme_io_qpair_opts opts;

	cdata = spdk_nvme_ctrlr_get_data(ctrlr);

	if (!spdk_nvme_ns_is_active(ns)) {
		printf("Controller %-20.20s (%-20.20s): Skipping inactive NS %u\n",
		       cdata->mn, cdata->sn,
		       spdk_nvme_ns_get_id(ns));
		g_warn = true;
		return;
	}

	ns_size = spdk_nvme_ns_get_size(ns);
	sector_size = spdk_nvme_ns_get_sector_size(ns);

	if (ns_size < g_io_size_bytes || sector_size > g_io_size_bytes) {
		printf("WARNING: controller %-20.20s (%-20.20s) ns %u has invalid "
		       "ns size %" PRIu64 " / block size %u for I/O size %u\n",
		       cdata->mn, cdata->sn, spdk_nvme_ns_get_id(ns),
		       ns_size, spdk_nvme_ns_get_sector_size(ns), g_io_size_bytes);
		g_warn = true;
		return;
	}

	max_xfer_size = spdk_nvme_ns_get_max_io_xfer_size(ns);
	spdk_nvme_ctrlr_get_default_io_qpair_opts(ctrlr, &opts, sizeof(opts));
	/* NVMe driver may add additional entries based on
	 * stripe size and maximum transfer size, we assume
	 * 1 more entry be used for stripe.
	 */
	entries = (g_io_size_bytes - 1) / max_xfer_size + 2;
	if ((g_queue_depth * entries) > opts.io_queue_size) {
		printf("controller IO queue size %u less than required\n",
		       opts.io_queue_size);
		printf("Consider using lower queue depth or small IO size because "
		       "IO requests may be queued at the NVMe driver.\n");
	}
	/* For requests which have children requests, parent request itself
	 * will also occupy 1 entry.
	 */
	entries += 1;

	entry = (struct ns_entry *)calloc(1, sizeof(struct ns_entry));
	if (entry == NULL) {
		perror("ns_entry malloc");
		exit(1);
	}

	entry->type = ENTRY_TYPE_NVME_NS;
	entry->u.nvme.ctrlr = ctrlr;
	entry->u.nvme.ns = ns;
	entry->num_io_requests = g_queue_depth * entries;

	entry->size_in_ios = ns_size / g_io_size_bytes;
	entry->io_size_blocks = g_io_size_bytes / sector_size;

	entry->block_size = spdk_nvme_ns_get_extended_sector_size(ns);
	entry->md_size = spdk_nvme_ns_get_md_size(ns);
	entry->md_interleave = spdk_nvme_ns_supports_extended_lba(ns);
	entry->pi_loc = spdk_nvme_ns_get_data(ns)->dps.md_start;
	entry->pi_type = spdk_nvme_ns_get_pi_type(ns);

	if (spdk_nvme_ns_get_flags(ns) & SPDK_NVME_NS_DPS_PI_SUPPORTED) {
		entry->io_flags = g_metacfg_pract_flag | g_metacfg_prchk_flags;
	}

	/* If metadata size = 8 bytes, PI is stripped (read) or inserted (write),
	 *  and so reduce metadata size from block size.  (If metadata size > 8 bytes,
	 *  PI is passed (read) or replaced (write).  So block size is not necessary
	 *  to change.)
	 */
	if ((entry->io_flags & SPDK_NVME_IO_FLAGS_PRACT) && (entry->md_size == 8)) {
		entry->block_size = spdk_nvme_ns_get_sector_size(ns);
	}

	if (g_max_io_md_size < entry->md_size) {
		g_max_io_md_size = entry->md_size;
	}

	if (g_max_io_size_blocks < entry->io_size_blocks) {
		g_max_io_size_blocks = entry->io_size_blocks;
	}

	build_nvme_name(entry->name, sizeof(entry->name), ctrlr);

	g_num_namespaces++;
	entry->next = g_namespaces;
	g_namespaces = entry;
}

static void
unregister_namespaces(void)
{
	struct ns_entry *entry = g_namespaces;

	while (entry) {
		struct ns_entry *next = entry->next;
		free(entry);
		entry = next;
	}
}


static void
register_ctrlr(struct spdk_nvme_ctrlr *ctrlr, struct trid_entry *trid_entry)
{
	struct spdk_nvme_ns *ns;
	struct ctrlr_entry *entry = (struct ctrlr_entry *)malloc(sizeof(struct ctrlr_entry));
	uint32_t nsid;

	if (entry == NULL) {
		perror("ctrlr_entry malloc");
		exit(1);
	}

	entry->latency_page = (struct spdk_nvme_intel_rw_latency_page *)spdk_dma_zmalloc(sizeof(struct spdk_nvme_intel_rw_latency_page),
					       4096, NULL);
	if (entry->latency_page == NULL) {
		printf("Allocation error (latency page)\n");
		exit(1);
	}

	build_nvme_name(entry->name, sizeof(entry->name), ctrlr);

	entry->ctrlr = ctrlr;
	entry->trtype = trid_entry->trid.trtype;
	entry->next = g_controllers;
	g_controllers = entry;

	if (trid_entry->nsid == 0) {
		for (nsid = spdk_nvme_ctrlr_get_first_active_ns(ctrlr);
		     nsid != 0; nsid = spdk_nvme_ctrlr_get_next_active_ns(ctrlr, nsid)) {
			ns = spdk_nvme_ctrlr_get_ns(ctrlr, nsid);
			if (ns == NULL) {
				continue;
			}
			register_ns(ctrlr, ns);
		}
	} else {
		ns = spdk_nvme_ctrlr_get_ns(ctrlr, trid_entry->nsid);
		if (!ns) {
			perror("Namespace does not exist.");
			exit(1);
		}

		register_ns(ctrlr, ns);
	}

	if (g_nr_unused_io_queues) {
		int i;

		printf("Creating %u unused qpairs for controller %s\n", g_nr_unused_io_queues, entry->name);

		entry->unused_qpairs = (struct spdk_nvme_qpair **)calloc(g_nr_unused_io_queues, sizeof(struct spdk_nvme_qpair *));
		if (!entry->unused_qpairs) {
			fprintf(stderr, "Unable to allocate memory for qpair array\n");
			exit(1);
		}

		for (i = 0; i < g_nr_unused_io_queues; i++) {
			entry->unused_qpairs[i] = spdk_nvme_ctrlr_alloc_io_qpair(ctrlr, NULL, 0);
			if (!entry->unused_qpairs[i]) {
				fprintf(stderr, "Unable to allocate unused qpair. Did you request too many?\n");
				exit(1);
			}
		}
	}

}

static __thread unsigned int seed = 0;


static void usage(char *program_name)
{
	printf("%s options", program_name);
#if HAVE_LIBAIO
	printf(" [AIO device(s)]...");
#endif
	printf("\n");
	printf("\t[-q io depth]\n");
	printf("\t[-o io size in bytes]\n");
	printf("\t[-n number of io queues per namespace. default: 1]\n");
	printf("\t[-U number of unused io queues per controller. default: 0]\n");
	printf("\t[-w io pattern type, must be one of\n");
	printf("\t\t(read, write, randread, randwrite, rw, randrw)]\n");
	printf("\t[-M rwmixread (100 for reads, 0 for writes)]\n");
	printf("\t[-L enable latency tracking via sw, default: disabled]\n");
	printf("\t\t-L for latency summary, -LL for detailed histogram\n");
	printf("\t[-l enable latency tracking via ssd (if supported), default: disabled]\n");
	printf("\t[-t time in seconds]\n");
	printf("\t[-c core mask for I/O submission/completion.]\n");
	printf("\t\t(default: 1)\n");
	printf("\t[-D disable submission queue in controller memory buffer, default: enabled]\n");
	printf("\t[-H enable header digest for TCP transport, default: disabled]\n");
	printf("\t[-I enable data digest for TCP transport, default: disabled]\n");
	printf("\t[-N no shutdown notification process for controllers, default: disabled]\n");
	printf("\t[-r Transport ID for local PCIe NVMe or NVMeoF]\n");
	printf("\t Format: 'key:value [key:value] ...'\n");
	printf("\t Keys:\n");
	printf("\t  trtype      Transport type (e.g. PCIe, RDMA)\n");
	printf("\t  adrfam      Address family (e.g. IPv4, IPv6)\n");
	printf("\t  traddr      Transport address (e.g. 0000:04:00.0 for PCIe or 192.168.100.8 for RDMA)\n");
	printf("\t  trsvcid     Transport service identifier (e.g. 4420)\n");
	printf("\t  subnqn      Subsystem NQN (default: %s)\n", SPDK_NVMF_DISCOVERY_NQN);
	printf("\t Example: -r 'trtype:PCIe traddr:0000:04:00.0' for PCIe or\n");
	printf("\t          -r 'trtype:RDMA adrfam:IPv4 traddr:192.168.100.8 trsvcid:4420' for NVMeoF\n");
	printf("\t[-e metadata configuration]\n");
	printf("\t Keys:\n");
	printf("\t  PRACT      Protection Information Action bit (PRACT=1 or PRACT=0)\n");
	printf("\t  PRCHK      Control of Protection Information Checking (PRCHK=GUARD|REFTAG|APPTAG)\n");
	printf("\t Example: -e 'PRACT=0,PRCHK=GUARD|REFTAG|APPTAG'\n");
	printf("\t          -e 'PRACT=1,PRCHK=GUARD'\n");
	printf("\t[-k keep alive timeout period in millisecond]\n");
	printf("\t[-s DPDK huge memory size in MB.]\n");
	printf("\t[-m max completions per poll]\n");
	printf("\t\t(default: 0 - unlimited)\n");
	printf("\t[-i shared memory group ID]\n");
	printf("\t");
	spdk_log_usage(stdout, "-T");
	printf("\t[-V enable VMD enumeration]\n");
#ifdef DEBUG
	printf("\t[-G enable debug logging]\n");
#else
	printf("\t[-G enable debug logging (flag disabled, must reconfigure with --enable-debug)\n");
#endif
}


static void
unregister_trids(void)
{
	struct trid_entry *trid_entry, *tmp;

	TAILQ_FOREACH_SAFE(trid_entry, &g_trid_list, tailq, tmp) {
		TAILQ_REMOVE(&g_trid_list, trid_entry, tailq);
		free(trid_entry);
	}
}

static int
add_trid(const char *trid_str)
{
	struct trid_entry *trid_entry;
	struct spdk_nvme_transport_id *trid;
	const char *ns;

	trid_entry = (struct trid_entry *)calloc(1, sizeof(*trid_entry));
	if (trid_entry == NULL) {
		return -1;
	}

	trid = &trid_entry->trid;
	trid->trtype = SPDK_NVME_TRANSPORT_PCIE;
	snprintf(trid->subnqn, sizeof(trid->subnqn), "%s", SPDK_NVMF_DISCOVERY_NQN);

	if (spdk_nvme_transport_id_parse(trid, trid_str) != 0) {
		fprintf(stderr, "Invalid transport ID format '%s'\n", trid_str);
		free(trid_entry);
		return 1;
	}

	ns = strcasestr(trid_str, "ns:");
	if (ns) {
		char nsid_str[6]; /* 5 digits maximum in an nsid */
		int len;
		int nsid;

		ns += 3;

		len = strcspn(ns, " \t\n");
		if (len > 5) {
			fprintf(stderr, "NVMe namespace IDs must be 5 digits or less\n");
			free(trid_entry);
			return 1;
		}

		memcpy(nsid_str, ns, len);
		nsid_str[len] = '\0';

		nsid = spdk_strtol(nsid_str, 10);
		if (nsid <= 0 || nsid > 65535) {
			fprintf(stderr, "NVMe namespace IDs must be less than 65536 and greater than 0\n");
			free(trid_entry);
			return 1;
		}

		trid_entry->nsid = (uint16_t)nsid;
	}

	TAILQ_INSERT_TAIL(&g_trid_list, trid_entry, tailq);
	return 0;
}

static size_t
parse_next_key(const char **str, char *key, char *val, size_t key_buf_size,
	       size_t val_buf_size)
{
	const char *sep;
	const char *separator = ", \t\n";
	size_t key_len, val_len;

	*str += strspn(*str, separator);

	sep = strchr(*str, '=');
	if (!sep) {
		fprintf(stderr, "Key without '=' separator\n");
		return 0;
	}

	key_len = sep - *str;
	if (key_len >= key_buf_size) {
		fprintf(stderr, "Key length %zu is greater than maximum allowed %zu\n",
			key_len, key_buf_size - 1);
		return 0;
	}

	memcpy(key, *str, key_len);
	key[key_len] = '\0';

	*str += key_len + 1;	/* Skip key */
	val_len = strcspn(*str, separator);
	if (val_len == 0) {
		fprintf(stderr, "Key without value\n");
		return 0;
	}

	if (val_len >= val_buf_size) {
		fprintf(stderr, "Value length %zu is greater than maximum allowed %zu\n",
			val_len, val_buf_size - 1);
		return 0;
	}

	memcpy(val, *str, val_len);
	val[val_len] = '\0';

	*str += val_len;

	return val_len;
}

static int
parse_metadata(const char *metacfg_str)
{
	const char *str;
	size_t val_len;
	char key[32];
	char val[1024];

	if (metacfg_str == NULL) {
		return -EINVAL;
	}

	str = metacfg_str;

	while (*str != '\0') {
		val_len = parse_next_key(&str, key, val, sizeof(key), sizeof(val));
		if (val_len == 0) {
			fprintf(stderr, "Failed to parse metadata\n");
			return -EINVAL;
		}

		if (strcmp(key, "PRACT") == 0) {
			if (*val == '1') {
				g_metacfg_pract_flag = SPDK_NVME_IO_FLAGS_PRACT;
			}
		} else if (strcmp(key, "PRCHK") == 0) {
			if (strstr(val, "GUARD") != NULL) {
				g_metacfg_prchk_flags |= SPDK_NVME_IO_FLAGS_PRCHK_GUARD;
			}
			if (strstr(val, "REFTAG") != NULL) {
				g_metacfg_prchk_flags |= SPDK_NVME_IO_FLAGS_PRCHK_REFTAG;
			}
			if (strstr(val, "APPTAG") != NULL) {
				g_metacfg_prchk_flags |= SPDK_NVME_IO_FLAGS_PRCHK_APPTAG;
			}
		} else {
			fprintf(stderr, "Unknown key '%s'\n", key);
		}
	}

	return 0;
}

static int
parse_args(int argc, char **argv)
{
	const char *workload_type;
	int op;
	bool mix_specified = false;
	long int val;
	int rc;

	/* default value */
	g_queue_depth = 0;
	g_io_size_bytes = 0;
	workload_type = NULL;
	g_time_in_sec = 0;
	g_rw_percentage = -1;
	g_core_mask = NULL;
	g_max_completions = 0;

	while ((op = getopt(argc, argv, "c:e:i:lm:n:o:q:r:k:s:t:w:DGHILM:NT:U:V")) != -1) {
		switch (op) {
		case 'i':
		case 'm':
		case 'n':
		case 'o':
		case 'q':
		case 'k':
		case 's':
		case 't':
		case 'M':
		case 'U':
			val = spdk_strtol(optarg, 10);
			if (val < 0) {
				fprintf(stderr, "Converting a string to integer failed\n");
				return val;
			}
			switch (op) {
			case 'i':
				g_shm_id = val;
				break;
			case 'm':
				g_max_completions = val;
				break;
			case 'n':
				g_nr_io_queues_per_ns = val;
				break;
			case 'o':
				g_io_size_bytes = val;
				break;
			case 'q':
				g_queue_depth = val;
				break;
			case 'k':
				g_keep_alive_timeout_in_ms = val;
				break;
			case 's':
				g_dpdk_mem = val;
				break;
			case 't':
				g_time_in_sec = val;
				break;
			case 'M':
				g_rw_percentage = val;
				mix_specified = true;
				break;
			case 'U':
				g_nr_unused_io_queues = val;
				break;
			}
			break;
		case 'c':
			g_core_mask = optarg;
			break;
		case 'e':
			if (parse_metadata(optarg)) {
				usage(argv[0]);
				return 1;
			}
			break;
		case 'l':
			g_latency_ssd_tracking_enable = true;
			break;
		case 'r':
			if (add_trid(optarg)) {
				usage(argv[0]);
				return 1;
			}
			break;
		case 'w':
			workload_type = optarg;
			break;
		case 'D':
			g_disable_sq_cmb = 1;
			break;
		case 'G':
#ifndef DEBUG
			fprintf(stderr, "%s must be configured with --enable-debug for -G flag\n",
				argv[0]);
			usage(argv[0]);
			return 1;
#else
			spdk_log_set_flag("nvme");
			spdk_log_set_print_level(SPDK_LOG_DEBUG);
			break;
#endif
		case 'H':
			g_header_digest = 1;
			break;
		case 'I':
			g_data_digest = 1;
			break;
		case 'L':
			g_latency_sw_tracking_level++;
			break;
		case 'N':
			g_no_shn_notification = true;
			break;
		case 'T':
			rc = spdk_log_set_flag(optarg);
			if (rc < 0) {
				fprintf(stderr, "unknown flag\n");
				usage(argv[0]);
				exit(EXIT_FAILURE);
			}
			spdk_log_set_print_level(SPDK_LOG_DEBUG);
#ifndef DEBUG
			fprintf(stderr, "%s must be rebuilt with CONFIG_DEBUG=y for -T flag.\n",
				argv[0]);
			usage(argv[0]);
			return 0;
#endif
			break;
		case 'V':
			g_vmd = true;
			break;
		default:
			usage(argv[0]);
			return 1;
		}
	}

	if (!g_nr_io_queues_per_ns) {
		usage(argv[0]);
		return 1;
	}

	if (!g_queue_depth) {
		usage(argv[0]);
		return 1;
	}
	if (!g_io_size_bytes) {
		usage(argv[0]);
		return 1;
	}
	if (!workload_type) {
		usage(argv[0]);
		return 1;
	}
	if (!g_time_in_sec) {
		usage(argv[0]);
		return 1;
	}

	if (strcmp(workload_type, "read") &&
	    strcmp(workload_type, "write") &&
	    strcmp(workload_type, "randread") &&
	    strcmp(workload_type, "randwrite") &&
	    strcmp(workload_type, "rw") &&
	    strcmp(workload_type, "randrw")) {
		fprintf(stderr,
			"io pattern type must be one of\n"
			"(read, write, randread, randwrite, rw, randrw)\n");
		return 1;
	}

	if (!strcmp(workload_type, "read") ||
	    !strcmp(workload_type, "randread")) {
		g_rw_percentage = 100;
	}

	if (!strcmp(workload_type, "write") ||
	    !strcmp(workload_type, "randwrite")) {
		g_rw_percentage = 0;
	}

	if (!strcmp(workload_type, "read") ||
	    !strcmp(workload_type, "randread") ||
	    !strcmp(workload_type, "write") ||
	    !strcmp(workload_type, "randwrite")) {
		if (mix_specified) {
			fprintf(stderr, "Ignoring -M option... Please use -M option"
				" only when using rw or randrw.\n");
		}
	}

	if (!strcmp(workload_type, "rw") ||
	    !strcmp(workload_type, "randrw")) {
		if (g_rw_percentage < 0 || g_rw_percentage > 100) {
			fprintf(stderr,
				"-M must be specified to value from 0 to 100 "
				"for rw or randrw.\n");
			return 1;
		}
	}

	if (!strcmp(workload_type, "read") ||
	    !strcmp(workload_type, "write") ||
	    !strcmp(workload_type, "rw")) {
		g_is_random = 0;
	} else {
		g_is_random = 1;
	}

	if (TAILQ_EMPTY(&g_trid_list)) {
		/* If no transport IDs specified, default to enumerating all local PCIe devices */
		add_trid("trtype:PCIe");
	} else {
		struct trid_entry *trid_entry, *trid_entry_tmp;

		g_no_pci = true;
		/* check whether there is local PCIe type */
		TAILQ_FOREACH_SAFE(trid_entry, &g_trid_list, tailq, trid_entry_tmp) {
			if (trid_entry->trid.trtype == SPDK_NVME_TRANSPORT_PCIE) {
				g_no_pci = false;
				break;
			}
		}
	}

	g_aio_optind = optind;

	return 0;
}

static bool
probe_cb(void *cb_ctx, const struct spdk_nvme_transport_id *trid,
	 struct spdk_nvme_ctrlr_opts *opts)
{
	if (trid->trtype == SPDK_NVME_TRANSPORT_PCIE) {
		if (g_disable_sq_cmb) {
			opts->use_cmb_sqs = false;
		}
		if (g_no_shn_notification) {
			opts->no_shn_notification = true;
		}
	}

	/* Set io_queue_size to UINT16_MAX, NVMe driver
	 * will then reduce this to MQES to maximize
	 * the io_queue_size as much as possible.
	 */
	opts->io_queue_size = UINT16_MAX;

	/* Set the header and data_digest */
	opts->header_digest = g_header_digest;
	opts->data_digest = g_data_digest;
	opts->keep_alive_timeout_ms = spdk_max(opts->keep_alive_timeout_ms,
					       g_keep_alive_timeout_in_ms);

	return true;
}

static void
attach_cb(void *cb_ctx, const struct spdk_nvme_transport_id *trid,
	  struct spdk_nvme_ctrlr *ctrlr, const struct spdk_nvme_ctrlr_opts *opts)
{
	struct trid_entry	*trid_entry = (struct trid_entry *)cb_ctx;
	struct spdk_pci_addr	pci_addr;
	struct spdk_pci_device	*pci_dev;
	struct spdk_pci_id	pci_id;

	if (trid->trtype != SPDK_NVME_TRANSPORT_PCIE) {
		printf("Attached to NVMe over Fabrics controller at %s:%s: %s\n",
		       trid->traddr, trid->trsvcid,
		       trid->subnqn);
	} else {
		if (spdk_pci_addr_parse(&pci_addr, trid->traddr)) {
			return;
		}

		pci_dev = spdk_nvme_ctrlr_get_pci_device(ctrlr);
		if (!pci_dev) {
			return;
		}

		pci_id = spdk_pci_device_get_id(pci_dev);

		printf("Attached to NVMe Controller at %s [%04x:%04x]\n",
		       trid->traddr,
		       pci_id.vendor_id, pci_id.device_id);
	}

	register_ctrlr(ctrlr, trid_entry);
}

static int
register_controllers(void)
{
	struct trid_entry *trid_entry;

	printf("Initializing NVMe Controllers\n");

	if (g_vmd && spdk_vmd_init()) {
		fprintf(stderr, "Failed to initialize VMD."
			" Some NVMe devices can be unavailable.\n");
	}

	TAILQ_FOREACH(trid_entry, &g_trid_list, tailq) {
		if (spdk_nvme_probe(&trid_entry->trid, trid_entry, probe_cb, attach_cb, NULL) != 0) {
			fprintf(stderr, "spdk_nvme_probe() failed for transport address '%s'\n",
				trid_entry->trid.traddr);
			return -1;
		}
	}

	return 0;
}

static void
unregister_controllers(void)
{
	struct ctrlr_entry *entry = g_controllers;

	while (entry) {
		struct ctrlr_entry *next = entry->next;
		spdk_dma_free(entry->latency_page);

		if (g_nr_unused_io_queues) {
			int i;

			for (i = 0; i < g_nr_unused_io_queues; i++) {
				spdk_nvme_ctrlr_free_io_qpair(entry->unused_qpairs[i]);
			}

			free(entry->unused_qpairs);
		}

		spdk_nvme_detach(entry->ctrlr);
		free(entry);
		entry = next;
	}
}

static void *
nvme_poll_ctrlrs(void *arg)
{
	struct ctrlr_entry *entry;
	int oldstate;

	spdk_unaffinitize_thread();

	while (true) {
		pthread_setcancelstate(PTHREAD_CANCEL_DISABLE, &oldstate);

		entry = g_controllers;
		while (entry) {
			if (entry->trtype != SPDK_NVME_TRANSPORT_PCIE) {
				spdk_nvme_ctrlr_process_admin_completions(entry->ctrlr);
			}
			entry = entry->next;
		}

		pthread_setcancelstate(PTHREAD_CANCEL_ENABLE, &oldstate);

		/* This is a pthread cancellation point and cannot be removed. */
		sleep(1);
	}

	return NULL;
}

struct lba_ranges {
	uint32_t phys_lba_start;
	uint32_t lba_count;
};

struct nvme_file_reader_ctx;

struct nvme_file_read_io_ctx {
	struct nvme_file_reader_ctx* main;
	uint32_t idx;			//idx of this IO request
	uint32_t lba_count;		//count of lba's to be read in this request
};

struct nvme_file_reader_ctx {
	struct spdk_nvme_ctrlr*	ctrlr;
	struct spdk_nvme_ns*	ns;
	struct spdk_nvme_qpair*	qpair;

	uint32_t				qdepth;
	struct nvme_file_read_io_ctx* reqs;

	uint32_t 				max_lba_per_io;

	struct iovec			data;

	uint32_t				lba_ranges_count;
	struct lba_ranges*		lba_ranges;

	uint32_t				lba_array_idx;	//current idx in lba_ranges
	uint32_t 				lba_idx;		//current number of lba in lba_ranges
	uint32_t 				lba_count;		//total number of lba's
	uint32_t 				lba_read;		//number of lba already read
	uint32_t 				lba_submitted_to_read;	//number of lba's submitted to read but not completted yet

	uint32_t 				file_size;		//in bytes
	uint32_t 				dev_block_size;
};

int fill_lba_ranges(const char* filepath, struct nvme_file_reader_ctx* ctx)
{
	int flags, fd;
	int bs;
	uint32_t bs_log2;
	int rc = 0;

	flags = O_DIRECT | O_RDONLY;

	fd = open(filepath, flags);
	if (fd < 0) {
		fprintf(stderr, "Could not open AIO device %s: %s\n", filepath, strerror(errno));
		return -1;
	}

	ctx->file_size = spdk_fd_get_size(fd);
	printf("file size %lu\n", ctx->file_size);
	if (ctx->file_size == 0) {
		fprintf(stderr, "Could not determine size of AIO device %s\n", filepath);
		close(fd);
		return -1;
	}

	if (ioctl(fd, FIGETBSZ, &bs) < 0) {
		printf("failed to get block size\n");
		close(fd);
		return -1;
	}

	ctx->lba_count = (ctx->file_size + (ctx->dev_block_size - 1)) / ctx->dev_block_size;
	bs_log2 = spdk_u32log2(ctx->dev_block_size);
	printf("device block size %d (%u)\n", bs, bs_log2);
	printf("file uses %u blocks\n", ctx->lba_count);

	union { struct fiemap f; char c[4096]; } fiemap_buf;
	struct fiemap *fiemap = &fiemap_buf.f;
	struct fiemap_extent *fm_extents = &fiemap->fm_extents[0];
	enum { count = (sizeof fiemap_buf - sizeof (*fiemap))/sizeof (*fm_extents) };
	memset (&fiemap_buf, 0, sizeof fiemap_buf);

	fiemap->fm_extent_count = count;
	fiemap->fm_length = FIEMAP_MAX_OFFSET;

	rc = ioctl (fd, FS_IOC_FIEMAP, fiemap);
	if(rc) {
		printf("ioctl failed, rc %d errno %d\n", rc, errno);
		return rc;
	}

	printf("fiemap: extents %u fm_flags %u\n",
		   fiemap->fm_mapped_extents, fiemap->fm_flags);

	if(fiemap->fm_mapped_extents == 0) {
		fprintf(stderr, "fiemap contains 0 extents\n");
		return -1;
	}
	ctx->lba_ranges_count = fiemap->fm_mapped_extents;
	ctx->lba_ranges = (struct lba_ranges*)calloc(ctx->lba_ranges_count, sizeof(*ctx->lba_ranges));

	for(uint32_t j = 0; j < fiemap->fm_mapped_extents; j++) {
		uint32_t length = fm_extents[j].fe_length >> bs_log2;
		uint32_t logical_start = fm_extents[j].fe_logical >> bs_log2;
		uint32_t physical_start = fm_extents[j].fe_physical >> bs_log2;
		printf("[%4u]: logical %8u .. %8u \tphysical %8u .. %8u \tlen %8u\n",
			   j, logical_start, logical_start + length -1,
			   physical_start, physical_start + length - 1, length);

		ctx->lba_ranges[j].phys_lba_start = physical_start;
		ctx->lba_ranges[j].lba_count = length;
	}

	close(fd);

	return 0;
}

int file_read_submit_io(struct nvme_file_read_io_ctx* ctx);

void file_read_complete_io(void *ctx , const struct spdk_nvme_cpl *cpl)
{
	struct nvme_file_read_io_ctx* req_ctx = (struct nvme_file_read_io_ctx*)ctx;
	struct nvme_file_reader_ctx* read_ctx = req_ctx->main;
	if (spdk_unlikely(spdk_nvme_cpl_is_error(cpl))) {
		fprintf(stderr, "read completed with error (sct=%d, sc=%d)\n",
				cpl->status.sct, cpl->status.sc);
	}

//	printf("completed req %u with lbas %u. Total %u/%u\n", req_ctx->idx,
//			req_ctx->lba_count, read_ctx->lba_read, read_ctx->lba_count);
	read_ctx->lba_read += req_ctx->lba_count;
	assert(read_ctx->lba_read <= read_ctx->lba_count);

	if(read_ctx->lba_read < read_ctx->lba_count) {
		file_read_submit_io(req_ctx);
	}
}

int file_read_submit_io(struct nvme_file_read_io_ctx* req_ctx)
{
	struct nvme_file_reader_ctx* ctx = req_ctx->main;
	int rc = 0;
//	printf("submit start, read %u / %u, array %u / %u\n",
//		   ctx->lba_read, ctx->lba_count, ctx->lba_array_idx, ctx->lba_ranges_count);
	if (ctx->lba_read >= ctx->lba_count || ctx->lba_array_idx >= ctx->lba_ranges_count) {
//		printf("all io requests already issued\n");
		return rc;
	}

	uint32_t lba_count = spdk_min(ctx->lba_ranges[ctx->lba_array_idx].lba_count - ctx->lba_idx, ctx->max_lba_per_io);
	void* payload_ptr = ((char*)ctx->data.iov_base) + ctx->lba_submitted_to_read * ctx->dev_block_size;
	uint32_t start_lba = ctx->lba_ranges[ctx->lba_array_idx].phys_lba_start + ctx->lba_idx;
	ctx->lba_submitted_to_read += lba_count;

//	printf("iov_base %p, current %p; lba[%u] start %u, count %u - read %u/%u\n",
//		   ctx->data.iov_base, payload_ptr, ctx->lba_array_idx, start_lba, lba_count, ctx->lba_read, ctx->lba_count);

	ctx->lba_idx += lba_count;
	req_ctx->lba_count = lba_count;
	assert(ctx->lba_idx <= ctx->lba_ranges[ctx->lba_array_idx].lba_count);

	if(ctx->lba_idx  >= ctx->lba_ranges[ctx->lba_array_idx].lba_count) {
//		printf("range %u completed, switch to the next\n", ctx->lba_array_idx);
		ctx->lba_array_idx++;
		ctx->lba_idx = 0;
	}

	rc = spdk_nvme_ns_cmd_read_with_md(ctx->ns, ctx->qpair, payload_ptr, NULL, start_lba, lba_count, file_read_complete_io,
									   req_ctx, 0, 0, 0);
	if(rc) {
		fprintf(stderr,"nvme read failed with %d\n", rc);
	}

	return rc;
}

__global__
void print_gpu_mem(char* c, int n)
{
	for (int i = 0; i < n; i++) {
		printf("%c", c[i]);
	}
	printf("\n");
}

struct nvme_file_read_ibv {
	struct ibv_context* context;
	struct ibv_pd* pd;
	struct ibv_mr* mr;
};

struct nvme_file_read_ibv* g_perf_ibv;
int g_perf_ibv_num_contexts;

struct ibv_mr* perf_get_mr(struct ibv_pd *pd, void *buf, size_t* size)
{
	assert(g_perf_ibv);
//	printf("addr %p size %zu\n", buf, size);
	for (int i = 0; i < g_perf_ibv_num_contexts; i++) {
		if(g_perf_ibv[i].pd == pd) {
			if(g_perf_ibv[i].mr) {
//				printf("match %d\n", i);
				assert((char*)buf >= (char*)g_perf_ibv[i].mr->addr);
				int64_t available = (int64_t)g_perf_ibv[i].mr->length - ((char*)g_perf_ibv[i].mr->addr - (char*)buf);
				if (available < 0 || available < (int64_t)*size) {
					fprintf(stderr, "request %zu bytes, available %ld\n", *size, available);
					*size = 0;
					return NULL;
				}
				*size = (size_t)available;
				return g_perf_ibv[i].mr;
			}
		}
	}
//	assert(0);
	return NULL;
}

struct ibv_pd* perf_get_pd(const struct spdk_nvme_transport_id *trid,
							 struct ibv_context *verbs)
{
	assert(g_perf_ibv);
	printf("verbs %p\n", verbs);
	for(uint32_t i = 0; i < g_perf_ibv_num_contexts; i++) {
		if(g_perf_ibv[i].context == verbs) {
			printf("match at idx %u\n", i);
			return g_perf_ibv[i].pd;
		}
	}
	assert(0);
	return NULL;
}

struct spdk_nvme_rdma_hooks g_perf_hooks = {
		.get_ibv_pd = perf_get_pd,
		.get_rkey = NULL,
		.get_user_mr = perf_get_mr
};

int alloc_ctx_and_pd(struct nvme_file_read_ibv** ctx, int* num)
{
	struct ibv_context ** contexts = rdma_get_devices(num);
	if (contexts == NULL) {
		fprintf(stderr, "failed to retrieve ibv devices\n");
		return -1;
	}
	printf("got %u ibv devices\n", *num);
	*ctx = (struct nvme_file_read_ibv*)calloc(*num, sizeof(struct nvme_file_read_ibv));

	for (int i = 0; i < *num; i++) {
		(*ctx)[i].context = contexts[i];
		(*ctx)[i].pd = ibv_alloc_pd(contexts[i]);
		if (!(*ctx)[i].pd) {
			fprintf(stderr, "failed to alloc PD\n");
			return -1;
		}
	}

	rdma_free_devices(contexts);

	return 0;
}

void free_mr_and_pd(struct nvme_file_read_ibv * ctx, int num)
{
	if(ctx) {
		for (int i = 0; i < num; i++) {
			if(ctx[i].pd) {
				ibv_dealloc_pd(ctx[i].pd);
			}
			if(ctx[i].mr) {
				ibv_dereg_mr(ctx[i].mr);
			}
		}
	}

	free(ctx);
}

int main(int argc, char **argv)
{
	int rc;
	struct worker_thread *worker, *master_worker;
	struct spdk_env_opts opts;
	struct nvme_file_reader_ctx read_ctx = {};
	struct spdk_nvme_io_qpair_opts qopts;

	int num_ibv_devices;
	pthread_t thread_id = 0;
	hipError_t res;

	rc = parse_args(argc, argv);
	if (rc != 0) {
		return rc;
	}

	spdk_env_opts_init(&opts);
	opts.name = "perf";
	opts.shm_id = g_shm_id;
	if (g_core_mask) {
		opts.core_mask = g_core_mask;
	}

	if (g_dpdk_mem) {
		opts.mem_size = g_dpdk_mem;
	}
	if (g_no_pci) {
		opts.no_pci = g_no_pci;
	}
	if (spdk_env_init(&opts) < 0) {
		fprintf(stderr, "Unable to initialize SPDK env\n");
		rc = -1;
		goto cleanup;
	}

	g_tsc_rate = spdk_get_ticks_hz();

	////////////////////////////////////////////////////////

	if (alloc_ctx_and_pd(&g_perf_ibv, &g_perf_ibv_num_contexts) != 0) {
		fprintf(stderr, "failed to alloc PDs\n");
		rc = -1;
		goto cleanup;
	}

	spdk_nvme_rdma_init_hooks(&g_perf_hooks);

	if (register_controllers() != 0) {
		rc = -1;
		goto cleanup;
	}

	if (g_warn) {
		printf("WARNING: Some requested NVMe devices were skipped\n");
	}

	if (g_num_namespaces == 0) {
		fprintf(stderr, "No valid NVMe controllers or AIO devices found\n");
		goto cleanup;
	}

	rc = pthread_create(&thread_id, NULL, &nvme_poll_ctrlrs, NULL);
	if (rc != 0) {
		fprintf(stderr, "Unable to spawn a thread to poll admin queues.\n");
		goto cleanup;
	}

	read_ctx.ctrlr	= g_controllers->ctrlr;
	read_ctx.ns		= g_namespaces->u.nvme.ns;


	spdk_nvme_ctrlr_get_default_io_qpair_opts(read_ctx.ctrlr, &qopts, sizeof(qopts));
	qopts.io_queue_requests = g_queue_depth * 4;
	qopts.io_queue_size = g_queue_depth;
	read_ctx.qdepth = qopts.io_queue_size;
	printf("creating io qpair, depth %u num_requests %u\n", qopts.io_queue_size, qopts.io_queue_requests);

	read_ctx.qpair = spdk_nvme_ctrlr_alloc_io_qpair(read_ctx.ctrlr, &qopts, sizeof(qopts));
	if(!read_ctx.qpair) {
		fprintf(stderr, "failed to create IO qpair\n");
		rc = -1;
		goto cleanup;
	}

	read_ctx.reqs = (struct nvme_file_read_io_ctx*)calloc(read_ctx.qdepth, sizeof(*read_ctx.reqs));
	for (uint32_t i = 0; i < read_ctx.qdepth; i++) {
		read_ctx.reqs[i].main = &read_ctx;
		read_ctx.reqs[i].idx = i;
	}

	read_ctx.dev_block_size = spdk_nvme_ns_get_sector_size(read_ctx.ns);

	////////////////////////////////////////////////////////

	printf("treat argv[%d] = %s as a file name\n", g_aio_optind, argv[g_aio_optind]);

	rc = fill_lba_ranges(argv[g_aio_optind], &read_ctx);
	if (rc) {
		fprintf(stderr, "file parsing failed");
		goto cleanup;
	}
	////////////////////////////////////////////////////////

	read_ctx.max_lba_per_io = spdk_nvme_ctrlr_get_max_xfer_size(read_ctx.ctrlr) / read_ctx.dev_block_size;
	read_ctx.data.iov_len = read_ctx.lba_count * read_ctx.dev_block_size;

#ifdef CUDA_DRAM
	read_ctx.data.iov_base = spdk_dma_zmalloc(read_ctx.data.iov_len, g_io_align, NULL);

	res = hipHostRegister(read_ctx.data.iov_base, read_ctx.data.iov_len, hipHostRegisterDefault);
	if(res != hipSuccess) {
		fprintf(stderr, "hipHostRegister failed with %d\n", res);
		rc = -1;
		goto cleanup;
	}
#else
	 res = hipMalloc(&read_ctx.data.iov_base, read_ctx.data.iov_len);
	if (res != hipSuccess) {
		fprintf(stderr, "failed to allocate GPU memory\n");
		rc = -1;
		goto cleanup;
	}

#endif

	for (int i = 0; i < g_perf_ibv_num_contexts; i++) {
		g_perf_ibv[i].mr = ibv_reg_mr(g_perf_ibv[i].pd, read_ctx.data.iov_base, read_ctx.data.iov_len,
				IBV_ACCESS_REMOTE_WRITE | IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_READ);
		if (g_perf_ibv[i].mr == NULL) {
			fprintf(stderr, "failed to register MR, errno %d\n", errno);
			rc = -1;
			goto cleanup;
		}
	}

	for(uint32_t i = 0; i < read_ctx.qdepth; i++) {
		file_read_submit_io(&read_ctx.reqs[i]);
	}

	while (read_ctx.lba_read < read_ctx.lba_count)
	{
		int completions = spdk_nvme_qpair_process_completions(read_ctx.qpair, read_ctx.qdepth);
		if(completions < 0) {
			fprintf(stderr, "process_completions failed with %d\n", completions);
			exit(1);
		}
	}

	printf("Done!\n");

#ifdef CUDA_DRAM
	printf("CPU 150 symbols:\n\n");

	for(uint32_t i = 0; i < 150; i++) {
		printf("%c", ((char*)read_ctx.data.iov_base)[i]);
	}
	printf("\n");
#endif

	printf("Running kernel to print 150 symbols\n");
	print_gpu_mem<<<1, 1>>>((char*)read_ctx.data.iov_base, 150);

cleanup:
	if (thread_id && pthread_cancel(thread_id) == 0) {
		pthread_join(thread_id, NULL);
	}

	if (read_ctx.lba_ranges) {
		free(read_ctx.lba_ranges);
	}

	if (g_perf_ibv) {
		free_mr_and_pd(g_perf_ibv, g_perf_ibv_num_contexts);
	}

	if (read_ctx.data.iov_base) {
#ifdef CUDA_DRAM
		hipHostUnregister(read_ctx.data.iov_base);
		spdk_dma_free(read_ctx.data.iov_base);
#else
		hipFree(read_ctx.data.iov_base);
#endif
	}

	if(read_ctx.qpair) {
		spdk_nvme_ctrlr_free_io_qpair(read_ctx.qpair);
	}

	unregister_trids();
	unregister_namespaces();
	unregister_controllers();

	if (rc != 0) {
		fprintf(stderr, "%s: errors occured\n", argv[0]);
	}

	return rc;
}
